#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "pinnedmem.cuh"

hipError_t
mallocHost(void** h_mem ,uint memSize, memoryMode memMode, bool wc)
{
    if( PINNED == memMode ) {
#if CUDART_VERSION >= 2020
        return hipHostAlloc( h_mem, memSize, (wc) ? hipHostMallocWriteCombined : 0 );
#else
        if (wc) {printf("Write-Combined unavailable on CUDART_VERSION less than 2020, running is: %d", CUDART_VERSION);}
        return hipHostMalloc( h_mem, memSize );
#endif
    }
    else { // PAGEABLE memory mode
        *h_mem = malloc( memSize );
    }

    return hipSuccess;
}

hipError_t
freeHost(void* h_mem, memoryMode memMode)
{
    if( PINNED == memMode ) {
        return hipHostFree(h_mem);
    }
    else {
        free(h_mem);
    }
    return hipSuccess;
}

/*
 * exitOnError: Show the error message and terminate the application.
 */ 
void exitOnError(const char *whereAt) {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
        {
            // print the CUDA error message and exit
            printf("CUDA error at %s: %s\n", whereAt, hipGetErrorString(error));
            exit(-1);
        }
}
