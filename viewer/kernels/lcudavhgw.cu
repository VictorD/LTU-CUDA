#include "hip/hip_runtime.h"
#include "../sharedmem.cuh"

template <class dataType, morphOperation MOP>
__global__ void _horizontalVHGWKernel(const dataType *img, int imgStep, dataType *result, 
                                    int resultStep, unsigned int width, unsigned int height,
                                        unsigned int size, rect2d borderSize) {
	const unsigned int step   = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	const unsigned int y      = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const unsigned int startx = __umul24(step,size);

    if (y >= height || startx > width)
        return;

    const dataType *lineIn = img+y*imgStep;
    dataType *lineOut      = result+y*resultStep;
    const unsigned int center  = startx + (size-1);

    dataType minarray[512];
    minarray[size-1] = lineIn[center];

    dataType nextMin;
    unsigned int k;
    if (MOP == ERODE) {
        for(k=1;k<size; ++k) {
            nextMin = lineIn[center-k];
            minarray[size-1-k] = min(minarray[size-k], nextMin);

            nextMin = (center+k < width+size-1) ? lineIn[center+k] : 255;
            minarray[size-1+k] = min(minarray[size+k-2], nextMin);
        }
    } else {
        for(k=1;k<size; ++k) {
            nextMin = lineIn[__umul24(center-k,imgStep)];
            minarray[size-1-k] = max(minarray[size-k], nextMin);
        
            nextMin = lineIn[__umul24(center+k,imgStep)];
            minarray[size-1+k] = max(minarray[size+k-2], nextMin);
        }
    }

    int diff = width - startx;
    if (diff > 0) {
        lineOut += startx;
        lineOut[0] = minarray[0];

        for(k=1; k < size-1; ++k) {
            if (diff > k) {
                lineOut[k] = minMax<dataType, MOP>(minarray[k], minarray[k+size-1]);
            }
        }

        if (diff > size-1) {
            lineOut[size-1] = minarray[2*(size-1)];
        }
    }
}

template <class dataType, morphOperation MOP>
__global__ void _verticalVHGWKernel(const dataType *img, int imgStep, dataType *result, 
                                    int resultStep, unsigned int width, unsigned int height,
                                        unsigned int size, rect2d borderSize) {
	const unsigned int x      = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	const unsigned int step   = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    const unsigned int starty = __umul24(step,size);

    if (x >= width || starty > height)
        return;

    const dataType *lineIn = img+x;
    dataType *lineOut      = result+x;

    const unsigned int center  = starty + (size-1);

    dataType minarray[512];
    minarray[size-1] = lineIn[center*imgStep];

    dataType nextMin;
    unsigned int k;
    if (MOP == ERODE) {
        for(k=1;k<size; ++k) {
            nextMin = lineIn[(center-k)*imgStep];
            minarray[size-1-k] = min(minarray[size-k], nextMin);

            nextMin = (center+k < height+size-1) ? lineIn[(center+k)*imgStep] : 255;
            minarray[size-1+k] = min(minarray[size+k-2], nextMin);
        }
    } else {
        for(k=1;k<size; ++k) {
            nextMin = lineIn[__umul24(center-k,imgStep)];
            minarray[size-1-k] = max(minarray[size-k], nextMin);
        
            nextMin = lineIn[__umul24(center+k,imgStep)];
            minarray[size-1+k] = max(minarray[size+k-2], nextMin);
        }
    }

    int diff = height - starty;
    if (diff > 0) {
        lineOut += starty*resultStep;
        lineOut[0] = minarray[0];

        for(k=1; k < size-1; ++k) {
            if (diff > k) {
                lineOut[k*resultStep] = minMax<dataType, MOP>(minarray[k], minarray[k+size-1]);
            }
        }

        if (diff > size-1) {
            lineOut[(size-1)*resultStep] = minarray[2*(size-1)];
        }
    }
}

template <class dataType, morphOperation MOP, vhgwDirection DIRECTION>
int _globalVHGW(const dataType * img, int imgStep, dataType * result, 
                                        int resultStep, rect2d oSizeROI, unsigned int size, rect2d borderSize) {
    const unsigned int width = oSizeROI.width;
    const unsigned int height = oSizeROI.height;
    unsigned int steps;

    PRINTF("width %d, height %d\n", width, height);
    PRINTF("Border (w: %d , h: %d)\n", borderSize.width, borderSize.height);

    if (DIRECTION == VERTICAL) {
        steps = (height+size-1)/size;
        dim3 gridSize((width+128-1)/128, (steps+2-1)/2);
        dim3 blockSize(128, 2);
        _verticalVHGWKernel<dataType, MOP><<<gridSize,blockSize>>>(img, imgStep,result, resultStep, width, height, size, borderSize);
    } else { // HORIZONTAL
        steps = (width+size-1)/size;
        dim3 gridSize((steps+64-1)/64, (height+4-1)/4);
        dim3 blockSize(64, 4);
        _horizontalVHGWKernel<dataType, MOP><<<gridSize, blockSize, sizeof(dataType)*(width + size)>>>(img, imgStep,result, resultStep, width, height, size, borderSize);
    }

#if 1
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
       // exit(-1);
    }
#endif

    return LCUDA_SUCCESS;
}
