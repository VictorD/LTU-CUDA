#include "pinnedmem.cuh"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

hipError_t mallocHost(void** h_mem ,unsigned int memSize, memoryMode memMode, bool wc)
{
    if( PINNED == memMode ) {
#if CUDART_VERSION >= 2020
        return hipHostAlloc( h_mem, memSize, (wc) ? hipHostMallocWriteCombined : 0 );
#else
        if (wc) {printf("Write-Combined unavailable on CUDART_VERSION less than 2020, running is: %d", CUDART_VERSION);}
        return hipHostMalloc( h_mem, memSize );
#endif
    }
    else { // PAGEABLE memory mode
        *h_mem = malloc( memSize );
    }

    return hipSuccess;
}

hipError_t freeHost(void* h_mem, memoryMode memMode)
{
    if( PINNED == memMode ) {
        return hipHostFree(h_mem);
    }
    else {
        free(h_mem);
    }
    return hipSuccess;
}
